#include "hip/hip_runtime.h"
#include "hipError_t.h"
#include ""
#include <iostream>
#include "reduce.cuh"
#include "clustering_par_common.cuh"
#include "Timer.h"

__global__ void calcDistKernel(float* data, float* clusterData, int* ndk, float* dist)
{
    extern __shared__ float clusterData_shared[];
    int res_idx = blockDim.x * blockIdx.x + threadIdx.x;
    int n = ndk[0], d = ndk[1], k = ndk[2];
    int tid = threadIdx.x;

    // przepisz dane centr�w do pami�ci wsp�dzielonej
    if (tid < d * k)
        clusterData_shared[tid] = clusterData[tid];

    __syncthreads();

    int data_idx = res_idx / k;
    int c_idx = res_idx % k;
    if (res_idx < n * k)
    {
        float v = 0.f;
        // w p�tli kolejno obliczamy kwadrat r�nicy dla ka�dej wsp�rz�dnej
        // wynik zostawiamy w kwadracie dla optymalizacji
        for (int i = 0; i < d; i++)
        {
            float part_v = data[i * n + data_idx] - clusterData_shared[c_idx * d + i];
            v += part_v * part_v;
        }
        // zapisujemy wyniki w tablicy dist
        dist[res_idx] = v;
    }

    __syncthreads();
}

__global__ void updateIndexKernel(float* min_dist, int* ndk, int* indicies, int* delta)
{
    extern __shared__ float min_dist_shared[];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int n = ndk[0], d = ndk[1], k = ndk[2];

    int tid = threadIdx.x;

    // przepisujemy do pami�ci wsp�dzielonej odleg�o�ci mi�dzy centrami a punktami o indeksach: 
    // [blockIdx.x * blockDim.x, blockIdx.x * blockDim.x + blockDim.x - 1]
    for (int i = 0; i < k; i++)
    {
        int out_idx = blockDim.x * blockIdx.x * k + blockDim.x * i + tid;
        if (out_idx < n * k)
            min_dist_shared[blockDim.x * i + tid] = min_dist[out_idx];
        __syncthreads();

    }

    if (idx < n)
    {
        // znajdujemy najbli�szy klaster
        float min = min_dist_shared[tid * k];
        int min_idx = 0;
        int last_idx = indicies[idx];
        for (int i = 1; i < k; i++)
        {
            if (min > min_dist_shared[tid * k + i])
            {
                min = min_dist_shared[tid * k + i];
                min_idx = i;
            }
        }

        // je�eli centrum si� zmieni�o, zapisz nowe centrum i dodaj do delty
        if (min_idx != last_idx)
        {
            indicies[idx] = min_idx;
            delta[idx] = 1;
        }
    }

    __syncthreads();

}

int calculateNumThreads(int k)
{
    return std::max(32, 256 / (1 << (k - 1) / 48));
}

void Kmeans_par_UpdateClusterOnCPU(PointsData_SOA& data, int MAX_ITERS)
{
    // dane punkt�w
    float* dev_data;
    // dane centr�w klastr�w
    float* dev_clusterData;
    // parametry n, d, k
    int* dev_ndk;
    // tablica, w kt�rej zapisujemy 1, je�eli zmienili�my centrum dla danego punktu
    int* dev_delta;
    // tablica odleg�o�ci mi�dzy punktami a centrami
    float* dev_dist;
    // tablica indeks�w klastr�w dla punkt�w (do kt�rego klastra przynale�y dany punkt)
    int* dev_clusterIdx;


    int n = data.ndk[0], d = data.ndk[1], k = data.ndk[2];
    float* old_clusters = new float[d * k];
    int* clusterCount = new int[k];
    const int N_THREADS = calculateNumThreads(k);
    int delta = 0;
    int iter = 0;

    // obliczanie czasu dzialania poszczegolnych etapow
    float time_dist = 0;
    float time_index = 0;
    float time_update_clusters = 0;


    CALL(hipSetDevice(0));

    // malloc
    {
        Timer_CPU t("poczatkowa inicjalizacja danych CPU -> GPU", true);
        CALL(hipMalloc((void**)&dev_data, n * d * sizeof(float)));
        CALL(hipMalloc((void**)&dev_clusterData, d * k * sizeof(float)));
        CALL(hipMalloc((void**)&dev_ndk, 3 * sizeof(int)));
        CALL(hipMalloc((void**)&dev_delta, n * sizeof(int)));
        CALL(hipMalloc((void**)&dev_clusterIdx, n * sizeof(int)));
        CALL(hipMalloc((void**)&dev_dist, n * k * sizeof(float)));


        // memset/memcpy
        CALL(hipMemcpy(dev_data, data.data, n * d * sizeof(float), hipMemcpyHostToDevice));
        CALL(hipMemcpy(dev_ndk, data.ndk, 3 * sizeof(int), hipMemcpyHostToDevice));
    }


    do
    {
        iter++;
        memcpy(old_clusters, data.clusterData, d * k * sizeof(float));

        // inicjalizacja danych
        {
            Timer_CPU t("inicjalizacja danych CPU -> GPU");
            CALL(hipMemcpy(dev_clusterData, data.clusterData, d * k * sizeof(float), hipMemcpyHostToDevice));
            CALL(hipMemcpy(dev_clusterIdx, data.clusterIndex, n * sizeof(int), hipMemcpyHostToDevice));
            CALL(hipMemset(dev_dist, 0, n * k * sizeof(float)));
            CALL(hipMemset(dev_delta, 0, n * sizeof(int)));
        }

        dim3 threads = dim3(N_THREADS);
        dim3 calcBlocks = dim3((n * k + (N_THREADS - 1)) / N_THREADS);

        // obliczanie odleg�o�ci
        {
            Timer_CPU t("obliczanie odleglosci");
            calcDistKernel << <calcBlocks, threads, k* d * sizeof(float) >> > (dev_data, dev_clusterData, dev_ndk, dev_dist);
            CALL(hipGetLastError());
            CALL(hipDeviceSynchronize());
            time_dist += t.getElapsed();
        }


        dim3 updateBlocks = dim3((n + (N_THREADS - 1)) / N_THREADS);

        // obliczanie nowych indeks�w klastr�w dla punkt�w
        {
            Timer_CPU t("obliczanie nowych indeksow klastrow dla punktow");
            updateIndexKernel << <updateBlocks, threads, k* N_THREADS * sizeof(int) >> > (dev_dist, dev_ndk, dev_clusterIdx, dev_delta);
            CALL(hipGetLastError());
            CALL(hipDeviceSynchronize());
            time_index = t.getElapsed();
        }


        CALL(hipMemcpy(data.clusterIndex, dev_clusterIdx, n * sizeof(int), hipMemcpyDeviceToHost));

        // obliczanie delty
        {
            Timer_CPU("obliczanie delty");
            delta = reduce(dev_delta, n);
        }

        memset(data.clusterData, 0, sizeof(float) * k * d);
        memset(clusterCount, 0, sizeof(int) * k);

        // aktualizacja centr�w klastr�w
        {
            Timer_CPU t("aktualizacja centrow klastrow");
            for (int i = 0; i < n; i++)
            {
                clusterCount[data.clusterIndex[i]]++;
                for (int j = 0; j < d; j++)
                {
                    data.clusterData[data.clusterIndex[i] * d + j] += data.data[j * n + i];
                }
            }

            for (int i = 0; i < k; i++)
            {
                for (int j = 0; j < d; j++)
                {
                    if (clusterCount[i] > 0)
                        data.clusterData[i * d + j] /= clusterCount[i];
                    else data.clusterData[i * d + j] = old_clusters[i * d + j];
                }

            }
            time_update_clusters = t.getElapsed();
        }
    } while (delta > 0 && iter < MAX_ITERS);

    std::cout << "Ilosc iteracji: " << iter << ".\n";
    std::cout << "delta = " << delta << ".\n";
    std::cout << "Srednie obliczanie odleglosci od punktow do centrow klastrow: " << time_dist / iter << "s.\n";
    std::cout << "Srednie obliczanie nowych indeksow centrow dla punktow: " << time_index / iter << "s.\n";
    std::cout << "Srednie obliczanie nowych centrow klastrow: " << time_update_clusters / iter << "s.\n";


    // zwalnianie pami�ci
    hipFree(dev_clusterData);
    hipFree(dev_clusterIdx);
    hipFree(dev_data);
    hipFree(dev_delta);
    hipFree(dev_ndk);
    hipFree(dev_dist);

    delete old_clusters;
    delete clusterCount;
}
